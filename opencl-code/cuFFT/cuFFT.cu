#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

#define N (8192)

int main() {
    hipfftComplex* h_input  = (hipfftComplex*) malloc(sizeof(hipfftComplex)*N);
    hipfftComplex* h_output = (hipfftComplex*) malloc(sizeof(hipfftComplex)*N);

    for (int i = 0; i < N; i++) {
        h_input[i].x = (float)(i + 1); // real
        h_input[i].y = 0.0f;           // imag
    }

    hipfftComplex* d_data = NULL;
    hipMalloc((void**)&d_data, sizeof(hipfftComplex)*N);

    hipMemcpy(d_data, h_input, sizeof(hipfftComplex)*N, hipMemcpyHostToDevice);

    hipfftHandle plan;
    hipfftPlan1d(&plan, N, HIPFFT_C2C, 1);

    // Create CUDA events for timing.
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Record the start event.
    hipEventRecord(start, 0);

    // Execute the FFT kernel.
    hipfftExecC2C(plan, d_data, d_data, HIPFFT_FORWARD);

    // Record the stop event.
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    // Compute the elapsed time in milliseconds.
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Total kernel execution time: %0.3f ms\n", elapsedTime);

    // Cleanup the CUDA events.
    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipMemcpy(h_output, d_data, sizeof(hipfftComplex)*N, hipMemcpyDeviceToHost);

    printf("\nForward FFT:\n");
    for (int i = 0; i < 8; i++) {
        printf("  freq[%d] = (%f, %f)\n", i, h_output[i].x, h_output[i].y);
    }

    hipfftDestroy(plan);
    hipFree(d_data);
    free(h_input);
    free(h_output);

    printf("\nDone.\n");
    return 0;
}
